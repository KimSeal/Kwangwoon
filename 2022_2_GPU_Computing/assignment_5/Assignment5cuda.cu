#include "hip/hip_runtime.h"
%%writefile Assignment5cuda.cu

#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <chrono>         //header to calculate time
#include <iostream>

#define GRIDSIZE 16*1024 //16K
#define BLOCKSIZE 1024   //1K
#define TOTALSIZE (GRIDSIZE * BLOCKSIZE)

void genData(float* ptr, unsigned int size) { //put values
    while (size--) {
        *ptr++ = (float)(rand() % 1000) / 1000.0F;
    }
}

void adjDiff(float* dst, const float* src, unsigned int size) {  //function to calculate delay
    for (int i = 1; i < size; i++) {
        dst[i] = src[i] - src[i - 1];
    }
}

__global__ void adjDiff2(float* result, const float* input) {  //function to calculate delay
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0) {
        float x_i = input[i];
        float x_i_m1 = input[i - 1];
        result[i] = x_i - x_i_m1;
    }
}

__global__ void adjDiff3(float* result, float* input) {  //function to calculate delay
    __shared__ float s_data[BLOCKSIZE];
    unsigned int tx = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    s_data[tx] = input[i];
    __syncthreads();
    if (tx > 0) {
        result[i] = s_data[tx] - s_data[tx - 1];
    }
    else if (i > 0) {
        result[i] = s_data[tx] - input[i - 1];
    }
}

int main(void) {
    ////////////////////////////5-1(HOST version)
    float* pSource = NULL;   //variable array to genData & adjDiff function
    float* pResult = NULL;
    int i;

    pSource = (float*)malloc(TOTALSIZE * sizeof(float)); //put space to save
    pResult = (float*)malloc(TOTALSIZE * sizeof(float));

    genData(pSource, TOTALSIZE);  //generate input source data
    std::chrono::system_clock::time_point start = std::chrono::system_clock::now(); //take start time
    pResult[0] = 0.0F;                                                              //exceptional case for i=0
    adjDiff(pResult, pSource, TOTALSIZE);
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now();   //take end time
    std::chrono::nanoseconds duration_micro = end - start;
    printf("elapsed time = %lld usec \n", duration_micro / 1000);
    //print sample cases
    i = 1;
    printf("i=%d: %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE - 1;
    printf("i=%d: %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE / 2;
    printf("i=%d: %f=%f-%f\n\n", i, pResult[i], pSource[i], pSource[i - 1]);
    //free the memory
    free(pSource);
    free(pResult);
    //////////////////////////////////////5-2(CUDA global memory version)
    float* pSource2 = NULL;   //variable array to genData & adjDiff function
    float* pResult2 = NULL;
    int i2;

    pSource2 = (float*)malloc(TOTALSIZE * sizeof(float)); //put space to save
    pResult2 = (float*)malloc(TOTALSIZE * sizeof(float));

    genData(pSource2, TOTALSIZE);  //generate input source data

    float* pSourceDev2 = NULL;
    float* pResultDev2 = NULL;

    pResult2[0] = 0.0F;                                                              //exceptional case for i=0

    hipMalloc((void**)&pSourceDev2, TOTALSIZE * sizeof(float));
    hipMalloc((void**)&pResultDev2, TOTALSIZE * sizeof(float));

    //CUDA launch the kernel adjDiff
    hipMemcpy(pSourceDev2, pSource2, TOTALSIZE * sizeof(float), hipMemcpyHostToDevice);
    std::chrono::system_clock::time_point start2 = std::chrono::system_clock::now(); //take start time

    dim3 dimGrid2(GRIDSIZE, 1, 1);
    dim3 dimBlock2(BLOCKSIZE, 1, 1);
    adjDiff2 << <dimGrid2, dimBlock2 >> > (pResultDev2, pSourceDev2);

    std::chrono::system_clock::time_point end2 = std::chrono::system_clock::now();   //take end time
    hipMemcpy(pResult2, pResultDev2, TOTALSIZE * sizeof(float), hipMemcpyDeviceToHost);
    std::chrono::nanoseconds duration_micro2 = end2 - start2;
    printf("elapsed time = %lld usec \n", duration_micro2 / 1000);
    //print sample cases
    i2 = 1;
    printf("i=%d: %f=%f-%f\n", i2, pResult2[i2], pSource2[i2], pSource2[i2 - 1]);
    i2 = TOTALSIZE - 1;
    printf("i=%d: %f=%f-%f\n", i2, pResult2[i2], pSource2[i2], pSource2[i2 - 1]);
    i2 = TOTALSIZE / 2;
    printf("i=%d: %f=%f-%f\n\n", i2, pResult2[i2], pSource2[i2], pSource2[i2 - 1]);
    //free the memory
    free(pSource2);
    free(pResult2);
    hipFree(pSourceDev2);
    hipFree(pResultDev2);
    hipDeviceSynchronize();
    /////////////////////5-3(CUDA Shared memory version)
    float* pSource3 = NULL;   //variable array to genData & adjDiff function
    float* pResult3 = NULL;
    int i3;

    pSource3 = (float*)malloc(TOTALSIZE * sizeof(float)); //put space to save
    pResult3 = (float*)malloc(TOTALSIZE * sizeof(float));

    genData(pSource3, TOTALSIZE);  //generate input source data

    float* pSourceDev3 = NULL;
    float* pResultDev3 = NULL;

    pResult3[0] = 0.0F;                                                              //exceptional case for i=0

    hipMalloc((void**)&pSourceDev3, TOTALSIZE * sizeof(float));
    hipMalloc((void**)&pResultDev3, TOTALSIZE * sizeof(float));

    //CUDA launch the kernel adjDiff
    hipMemcpy(pSourceDev3, pSource3, TOTALSIZE * sizeof(float), hipMemcpyHostToDevice);
    std::chrono::system_clock::time_point start3 = std::chrono::system_clock::now(); //take start time

    dim3 dimGrid3(GRIDSIZE, 1, 1);
    dim3 dimBlock3(BLOCKSIZE, 1, 1);
    adjDiff3 << <dimGrid3, dimBlock3 >> > (pResultDev3, pSourceDev3);

    std::chrono::system_clock::time_point end3 = std::chrono::system_clock::now();   //take end time
    hipMemcpy(pResult3, pResultDev3, TOTALSIZE * sizeof(float), hipMemcpyDeviceToHost);
    std::chrono::nanoseconds duration_micro3 = end3 - start3;
    printf("elapsed time = %lld usec \n", duration_micro3 / 1000);
    //print sample cases
    i3 = 1;
    printf("i=%d: %f=%f-%f\n", i3, pResult3[i3], pSource3[i3], pSource3[i3 - 1]);
    i3 = TOTALSIZE - 1;
    printf("i=%d: %f=%f-%f\n", i3, pResult3[i3], pSource3[i3], pSource3[i3 - 1]);
    i3 = TOTALSIZE / 2;
    printf("i=%d: %f=%f-%f\n", i3, pResult3[i3], pSource3[i3], pSource3[i3 - 1]);
    //free the memory
    free(pSource3);
    free(pResult3);
    hipFree(pSourceDev3);
    hipFree(pResultDev3);
    hipDeviceSynchronize();
    return 0;
}
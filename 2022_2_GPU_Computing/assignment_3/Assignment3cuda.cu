#include "hip/hip_runtime.h"
﻿%%writefile Assignment3cuda.cu

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define WIDTH 5

__global__ void mulKernel(int* c, const int* a, const int* b) {
    int x = threadIdx.x;
    int y = threadIdx.y;
    int sum = 0;
    for (int i = 0; i < WIDTH; i++) {
        sum += a[y * WIDTH + i] * b[i * WIDTH + x];
    }
    c[WIDTH * y + x] = sum;

}

int main() {

    int a[WIDTH][WIDTH];
    int b[WIDTH][WIDTH];
    int c[WIDTH][WIDTH] = { 0 };
    for (int y = 0; y < WIDTH; y++) {
        for (int x = 0; x < WIDTH; x++) {
            a[y][x] = y * 10 + x;
            b[y][x] = (y * 10 + x);
        }
    }

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));

    hipMemcpy(dev_a, a, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);

    dim3 DimBlock(WIDTH, WIDTH);
    mulKernel << < 1, DimBlock >> > (dev_c, dev_a, dev_b); // launch test function
    hipMemcpy(c, dev_c, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("{%d} ", a[i][j]);    // print the results
        }
        for (int j = 0; j < WIDTH; j++) {
            printf("{%d} ", b[i][j]);    // print the results
        }
        for (int j = 0; j < WIDTH; j++) {
            printf("{%d} ", c[i][j]);    // print the results
        }
        printf("\n");
    }
    hipFree(dev_a);    // free the device memory spaces 
    hipFree(dev_b);    // free the device memory spaces 
    hipFree(dev_c);    // free the device memory spaces 
    return 0;
}
#include "hip/hip_runtime.h"
%% writefile Assignment4cuda.cu

#include <stdio.h>
#include <stdlib.h>
#include <iostream>



__global__ void mulKernel(int* c, const int* a, const int* b, const int WIDTH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    for (int i = 0; i < WIDTH; i++) {
        sum += a[y * WIDTH + i] * b[i * WIDTH + x];
    }
    c[WIDTH * y + x] = sum;

}

int main() {
    const int WIDTH = 16;
    const int TILE_WIDTH = 8;

    int a[WIDTH][WIDTH];
    int b[WIDTH][WIDTH];
    int c[WIDTH][WIDTH] = { 0 };
    for (int y = 0; y < WIDTH; y++) {
        for (int x = 0; x < WIDTH; x++) {
            a[y][x] = y * 10 + x;
            b[y][x] = (y * 10 + x);
        }
    }

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));

    hipMemcpy(dev_a, a, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);

    dim3 dimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH, 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    mulKernel << < dimGrid, DimBlock >> > (dev_c, dev_a, dev_b, WIDTH); // launch test function
    hipMemcpy(c, dev_c, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("[%6d] ", a[i][j]);    // print the results
            if (j % TILE_WIDTH == (TILE_WIDTH - 1)) {
                printf("   ");
            }
        }
        printf(" || ");
        for (int j = 0; j < WIDTH; j++) {
            printf("[%6d] ", b[i][j]);    // print the results
            if (j % TILE_WIDTH == (TILE_WIDTH - 1)) {
                printf("   ");
            }
        }
        printf(" || ");
        for (int j = 0; j < WIDTH; j++) {
            printf("[%6d] ", c[i][j]);    // print the results
            if (j % TILE_WIDTH == (TILE_WIDTH - 1)) {
                printf("   ");
            }
        }
        printf("\n");
        if (i % TILE_WIDTH == (TILE_WIDTH - 1)) {
            printf("\n");
        }
    }
    hipFree(dev_a);    // free the device memory spaces 
    hipFree(dev_b);    // free the device memory spaces 
    hipFree(dev_c);    // free the device memory spaces 
    return 0;
}
#include "hip/hip_runtime.h"
﻿%% writefile Assignment2cuda.cu
#include <stdlib.h>
#include <stdio.h>

__global__ void addKernel(int* d, const int* a, const int* b, const int* c) {
    int i = threadIdx.x;
    d[i] = a[i] + b[i] + c[i];
}

int main() {

    const int SIZE = 5;
    int a[SIZE] = { 0 };
    int b[SIZE] = { 0 };
    int c[SIZE] = { 0 };
    for (int i = 0; i < 5; i++) {
        a[i] = rand() / 4;
        b[i] = rand() / 4;
        c[i] = rand() / 4;
    }
    int d[SIZE] = { 0 };

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    int* dev_d = 0;

    hipMalloc((int**)&dev_a, SIZE * sizeof(int));
    hipMalloc((int**)&dev_b, SIZE * sizeof(int));
    hipMalloc((int**)&dev_c, SIZE * sizeof(int));
    hipMalloc((int**)&dev_d, SIZE * sizeof(int));

    hipMemcpy(dev_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    addKernel << < 1, SIZE >> > (dev_d, dev_a, dev_b, dev_c); // launch test function
    hipMemcpy(d, dev_d, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++)
        printf("{%d} + {%d} + {%d} = {%d}\n", a[i], b[i], c[i], d[i]);    // print the results

    hipFree(dev_a);    // free the device memory spaces 
    hipFree(dev_b);    // free the device memory spaces 
    hipFree(dev_c);    // free the device memory spaces 
    hipFree(dev_d);    // free the device memory spaces 
    return 0;
}